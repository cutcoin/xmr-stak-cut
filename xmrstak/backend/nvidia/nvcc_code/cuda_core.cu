#include "hip/hip_runtime.h"
#include "xmrstak/backend/cryptonight.hpp"

#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifdef _WIN32
#include <windows.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
    if (waitTime > 0)
    {
        if (waitTime > 100)
        {
            // use a waitable timer for larger intervals > 0.1ms

            HANDLE timer;
            LARGE_INTEGER ft;

            ft.QuadPart = -10ll * int64_t(waitTime); // Convert to 100 nanosecond interval, negative value indicates relative time

            timer = CreateWaitableTimer(NULL, TRUE, NULL);
            SetWaitableTimer(timer, &ft, 0, NULL, NULL, 0);
            WaitForSingleObject(timer, INFINITE);
            CloseHandle(timer);
        }
        else
        {
            // use a polling loop for short intervals <= 100ms

            LARGE_INTEGER perfCnt, start, now;
            __int64 elapsed;

            QueryPerformanceFrequency(&perfCnt);
            QueryPerformanceCounter(&start);
            do {
		SwitchToThread();
                QueryPerformanceCounter((LARGE_INTEGER*) &now);
                elapsed = (__int64)((now.QuadPart - start.QuadPart) / (float)perfCnt.QuadPart * 1000 * 1000);
            } while ( elapsed < waitTime );
        }
    }
}
#else
#include <unistd.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
	usleep(waitTime);
}
#endif

#include "cryptonight.hpp"
#include "cuda_extra.hpp"
#include "cuda_aes.hpp"
#include "cuda_device.hpp"

/* sm_2X is limited to 2GB due to the small TLB
 * therefore we never use 64bit indices
 */
#if defined(XMR_STAK_LARGEGRID) && (__CUDA_ARCH__ >= 300)
typedef uint64_t IndexType;
#else
typedef int IndexType;
#endif

__device__ __forceinline__ uint64_t cuda_mul128( uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi )
{
	*product_hi = __umul64hi( multiplier, multiplicand );
	return (multiplier * multiplicand );
}

template< typename T >
__device__ __forceinline__ T loadGlobal64( T * const addr )
{
	T x;
	asm volatile( "ld.global.cg.u64 %0, [%1];" : "=l"( x ) : "l"( addr ) );
	return x;
}

template< typename T >
__device__ __forceinline__ T loadGlobal32( T * const addr )
{
	T x;
	asm volatile( "ld.global.cg.u32 %0, [%1];" : "=r"( x ) : "l"( addr ) );
	return x;
}


template< typename T >
__device__ __forceinline__ void storeGlobal32( T* addr, T const & val )
{
	asm volatile( "st.global.cg.u32 [%0], %1;" : : "l"( addr ), "r"( val ) );
}

template<size_t ITERATIONS, uint32_t THREAD_SHIFT>
__global__ void cryptonight_core_gpu_phase1( int threads, int bfactor, int partidx, uint32_t * __restrict__ long_state, uint32_t * __restrict__ ctx_state, uint32_t * __restrict__ ctx_key1 )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );
	__syncthreads( );

	const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
	const int sub = ( threadIdx.x & 7 ) << 2;

	const int batchsize = ITERATIONS >> bfactor;
	const int start = partidx * batchsize;
	const int end = start + batchsize;

	if ( thread >= threads )
		return;

	uint32_t key[40], text[4];

	MEMCPY8( key, ctx_key1 + thread * 40, 20 );

	if( partidx == 0 )
	{
		// first round
		MEMCPY8( text, ctx_state + thread * 50 + sub + 16, 2 );
	}
	else
	{
		// load previous text data
		MEMCPY8( text, &long_state[( (uint64_t) thread << THREAD_SHIFT ) + sub + start - 32], 2 );
	}
	__syncthreads( );
	for ( int i = start; i < end; i += 32 )
	{
		cn_aes_pseudo_round_mut( sharedMemory, text, key );
		MEMCPY8(&long_state[((uint64_t) thread << THREAD_SHIFT) + (sub + i)], text, 2);
	}
}

/** avoid warning `unused parameter` */
template< typename T >
__forceinline__ __device__ void unusedVar( const T& )
{
}

/** shuffle data for
 *
 * - this method can be used with all compute architectures
 * - for <sm_30 shared memory is needed
 *
 * @param ptr pointer to shared memory, size must be `threadIdx.x * sizeof(uint32_t)`
 *            value can be NULL for compute architecture >=sm_30
 * @param sub thread number within the group, range [0;4)
 * @param value value to share with other threads within the group
 * @param src thread number within the group from where the data is read, range [0;4)
 */
__forceinline__ __device__ uint32_t shuffle(volatile uint32_t* ptr,const uint32_t sub,const int val,const uint32_t src)
{
#if( __CUDA_ARCH__ < 300 )
    ptr[sub] = val;
    return ptr[src&3];
#else
    unusedVar( ptr );
    unusedVar( sub );
#   if(__CUDACC_VER_MAJOR__ >= 9)
    return __shfl_sync(0xFFFFFFFF, val, src, 4 );
#	else
	return __shfl( val, src, 4 );
#	endif
#endif
}

template<size_t ITERATIONS, uint32_t THREAD_SHIFT, uint32_t MASK>
#ifdef XMR_STAK_THREADS
__launch_bounds__( XMR_STAK_THREADS * 4 )
#endif
__global__ void cryptonight_core_gpu_phase2( int threads, int bfactor, int partidx, uint32_t * d_long_state, uint32_t * d_ctx_a, uint32_t * d_ctx_b )
{
	__shared__ uint32_t sharedMemory[1024];
  extern __shared__ uint32_t shared_a[];
  uint32_t *shared_b = shared_a + blockDim.x;
  uint32_t *shared_c = shared_b + blockDim.x;

	cn_aes_gpu_init( sharedMemory );

	__syncthreads( );

	const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 2;
	const int sub = threadIdx.x & 3;

  if ( thread >= threads )
		return;

	int i;
  uint32_t al, bl, cl, tmpl, idx;
	const int batchsize = (ITERATIONS * 2) >> ( 1 + bfactor );
	const int start = partidx * batchsize;
	const int end = start + batchsize;
	uint32_t *long_state = &d_long_state[(IndexType) thread << THREAD_SHIFT];
	uint32_t *ctx_a = d_ctx_a + thread * 4;
	uint32_t *ctx_b = d_ctx_b + thread * 4;
  uint32_t *a = shared_a + ((threadIdx.x >> 2) << 2);
  uint32_t *b = shared_b + ((threadIdx.x >> 2) << 2);
  uint32_t *c = shared_c + ((threadIdx.x >> 2) << 2);

  a[sub] = ctx_a[sub];
  b[sub] = ctx_b[sub];

  for ( i = start; i < end; ++i )
  {
    __syncthreads( );
    idx = (a[0] & 0x1FFFC0) >> 2;
    cl = loadGlobal32<uint32_t>(long_state + idx + sub);
    cl = ROTL32(cl, b[0] & 31);
    cl += loadGlobal32<uint32_t>(long_state + idx + sub + 4);
    cl = ROTL32(cl, b[1] & 31);
    cl += loadGlobal32<uint32_t>(long_state + idx + sub + 8);
    cl = ROTL32(cl, b[2] & 31);
    cl += loadGlobal32<uint32_t>(long_state + idx + sub + 12);
    cl = ROTL32(cl, b[3] & 31);
    c[sub] = cl;

    __syncthreads( );

    cl = a[sub]  ^ (t_fn0(c[sub] & 0xff) ^ t_fn1((c[(sub + 1) & 3] >> 8) & 0xff) ^ t_fn2((c[(sub + 2) & 3] >> 16) & 0xff) ^ t_fn3((c[(sub + 3) & 3] >> 24)));

    c[sub] = cl;

    bl = b[sub];

    long_state[idx + sub] ^= ROTL32(bl, a[0] & 31) + cl;
    long_state[idx + sub + 4] ^= ROTL32(bl, a[1] & 31) + cl;
    long_state[idx + sub + 8] ^= ROTL32(bl, a[2] & 31) + cl;
    long_state[idx + sub + 12] ^= ROTL32(bl, a[3] & 31) + cl;

    __syncthreads( );

    idx = (c[0] & 0x1FFFC0) >> 2;
    tmpl = loadGlobal32<uint32_t>(long_state + idx + sub);
    tmpl = ROTL32(tmpl, a[0] & 31);
    tmpl -= loadGlobal32<uint32_t>(long_state + idx + sub + 4);
    tmpl = ROTL32(tmpl, a[1] & 31);
    tmpl -= loadGlobal32<uint32_t>(long_state + idx + sub + 8);
    tmpl = ROTL32(tmpl, a[2] & 31);
    tmpl -= loadGlobal32<uint32_t>(long_state + idx + sub + 12);
    tmpl = ROTL32(tmpl, a[3] & 31);

    al = a[sub];
    al += cl * tmpl;
    a[sub] = tmpl;

    __syncthreads( );

    long_state[idx + sub] ^= ROTL32(al, a[0] & 31);
    long_state[idx + sub + 4] ^= ROTL32(al, a[1] & 31);
    long_state[idx + sub + 8] ^= ROTL32(al, a[2] & 31);
    long_state[idx + sub + 12] ^= ROTL32(al, a[3] & 31);

     a[sub] ^= al;
     b[sub] = cl;
  }

  if ( bfactor > 0 )
  {
    ctx_a[sub] = a[sub];
    ctx_b[sub] = b[sub];
  }

}

template<size_t ITERATIONS, uint32_t THREAD_SHIFT>
__global__ void cryptonight_core_gpu_phase3( int threads, int bfactor, int partidx, const uint32_t * __restrict__ long_state, uint32_t * __restrict__ d_ctx_state, uint32_t * __restrict__ d_ctx_key2 )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );
	__syncthreads( );

	int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
	int sub = ( threadIdx.x & 7 ) << 2;

	const int batchsize = ITERATIONS >> bfactor;
	const int start = partidx * batchsize;
	const int end = start + batchsize;

	if ( thread >= threads )
		return;

	uint32_t key[40], text[4];
	MEMCPY8( key, d_ctx_key2 + thread * 40, 20 );
	MEMCPY8( text, d_ctx_state + thread * 50 + sub + 16, 2 );

	__syncthreads( );
	for ( int i = start; i < end; i += 32 )
	{
#pragma unroll
		for ( int j = 0; j < 4; ++j )
			text[j] ^= long_state[((IndexType) thread << THREAD_SHIFT) + (sub + i + j)];

		cn_aes_pseudo_round_mut( sharedMemory, text, key );
	}

	MEMCPY8( d_ctx_state + thread * 50 + sub + 16, text, 2 );
}

template<size_t ITERATIONS, uint32_t MASK, uint32_t THREAD_SHIFT>
void cryptonight_core_gpu_hash(nvid_ctx* ctx)
{
	dim3 grid( ctx->device_blocks );
	dim3 block( ctx->device_threads );
	dim3 block4( ctx->device_threads << 2 );
	dim3 block8( ctx->device_threads << 3 );

	int partcount = 1 << ctx->device_bfactor;

	/* bfactor for phase 1 and 3
	 *
	 * phase 1 and 3 consume less time than phase 2, therefore we begin with the
	 * kernel splitting if the user defined a `bfactor >= 5`
	 */
	int bfactorOneThree = ctx->device_bfactor - 4;
	if( bfactorOneThree < 0 )
		bfactorOneThree = 0;

	int partcountOneThree = 1 << bfactorOneThree;

	for ( int i = 0; i < partcountOneThree; i++ )
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase1<ITERATIONS,THREAD_SHIFT><<< grid, block8 >>>( ctx->device_blocks*ctx->device_threads,
			bfactorOneThree, i,
			ctx->d_long_state, ctx->d_ctx_state, ctx->d_ctx_key1 ));

		if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
	}
	if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );

	for ( int i = 0; i < partcount; i++ )
	{
        CUDA_CHECK_MSG_KERNEL(
			ctx->device_id,
			"\n**suggestion: Try to increase the value of the attribute 'bfactor' or \nreduce 'threads' in the NVIDIA config file.**",
			cryptonight_core_gpu_phase2<ITERATIONS,THREAD_SHIFT,MASK><<<
				grid,
				block4,
				block4.x * sizeof(uint32_t) * 3
			>>>(
				ctx->device_blocks*ctx->device_threads,
				ctx->device_bfactor,
				i,
				ctx->d_long_state,
				ctx->d_ctx_a,
				ctx->d_ctx_b
			)
	    );

		if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
	}

	for ( int i = 0; i < partcountOneThree; i++ )
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase3<ITERATIONS,THREAD_SHIFT><<< grid, block8 >>>( ctx->device_blocks*ctx->device_threads,
			bfactorOneThree, i,
			ctx->d_long_state,
			ctx->d_ctx_state, ctx->d_ctx_key2 ));
	}
}

void cryptonight_core_cpu_hash(nvid_ctx* ctx, bool mineMonero)
{
#ifndef CONF_NO_MONERO
	if(mineMonero)
	{
		cryptonight_core_gpu_hash<MONERO_ITER, MONERO_MASK, 19u>(ctx);
	}
#endif
#ifndef CONF_NO_AEON
	if(!mineMonero)
	{
		cryptonight_core_gpu_hash<AEON_ITER, AEON_MASK, 18u>(ctx);
	}
#endif
}
